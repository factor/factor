/*
 World using CUDA
** 
** The string "Hello World!" is mangled then restored using a common CUDA idiom
**
** Byron Galbraith
** 2009-02-18
*/
#include <hip/hip_runtime.h>
#include <stdio.h>

// Prototypes
extern "C" __global__ void helloWorld(char*);

// Host function
int
main(int argc, char** argv)
{
  int i;

  // desired output
  char str[] = "Hello World!";

  // mangle contents of output
  // the null character is left intact for simplicity
  for(i = 0; i < 12; i++)
    str[i] -= i;

  // allocate memory on the device 
  char *d_str;
  size_t size = sizeof(str);
  hipMalloc((void**)&d_str, size);

  // copy the string to the device
  hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

  // set the grid and block sizes
  dim3 dimGrid(2);   // one block per word  
  dim3 dimBlock(6); // one thread per character
  
  // invoke the kernel
  helloWorld<<< dimGrid, dimBlock >>>(d_str);

  // retrieve the results from the device
  hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

  // free up the allocated memory on the device
  hipFree(d_str);
  
  // everyone's favorite part
  printf("%s\n", str);

  return 0;
}

// Device kernel
__global__ void
helloWorld(char* str)
{
  // determine where in the thread grid we are
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // unmangle output
  str[idx] += idx;
}
