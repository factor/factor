#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

static const int LOG_BANK_COUNT = 4;

static inline __device__ __host__ unsigned shared_offset(unsigned i)
{
    return i + (i >> LOG_BANK_COUNT);
}

static inline __device__ __host__ unsigned offset_a(unsigned offset, unsigned i)
{
    return shared_offset(offset * (2*i + 1) - 1);
}

static inline __device__ __host__ unsigned offset_b(unsigned offset, unsigned i)
{
    return shared_offset(offset * (2*i + 2) - 1);
}

static inline __device__ __host__ unsigned lpot(unsigned x)
{
    --x; x |= x>>1; x|=x>>2; x|=x>>4; x|=x>>8; x|=x>>16; return ++x;
}

template<typename T>
__global__ void prefix_sum_block(T *in, T *out, unsigned n)
{
    extern __shared__ T temp[];

    int idx = threadIdx.x;
    int blocksize = blockDim.x;

    temp[shared_offset(idx            )] = (idx             < n) ? in[idx            ] : 0;
    temp[shared_offset(idx + blocksize)] = (idx + blocksize < n) ? in[idx + blocksize] : 0;

    int offset, d;
    for (offset = 1, d = blocksize; d > 0; d >>= 1, offset <<= 1) {
        __syncthreads();
        if (idx < d) {
            unsigned a = offset_a(offset, idx), b = offset_b(offset, idx);
            temp[b] += temp[a];
        }
    }

    if (idx == 0) temp[shared_offset(blocksize*2 - 1)] = 0;

    for (d = 1; d <= blocksize; d <<= 1) {
        offset >>= 1;
        __syncthreads();

        if (idx < d) {
            unsigned a = offset_a(offset, idx), b = offset_b(offset, idx);
            unsigned t = temp[a];
            temp[a] = temp[b];
            temp[b] += t;
        }
    }
    __syncthreads();

    if (idx             < n) out[idx            ] = temp[shared_offset(idx            )];
    if (idx + blocksize < n) out[idx + blocksize] = temp[shared_offset(idx + blocksize)];
}

template<typename T>
void prefix_sum(T *in, T *out, unsigned n)
{
    char *device_values;
    unsigned n_lpot = lpot(n);
    size_t n_pitch;

    hipError_t error = hipMallocPitch((void**)&device_values, &n_pitch, sizeof(T)*n, 2);
    if (error != 0) {
        printf("error %u allocating width %lu height %u\n", error, sizeof(T)*n, 2);
        exit(1);
    }

    hipMemcpy(device_values, in, sizeof(T)*n, hipMemcpyHostToDevice);

    prefix_sum_block<<<1, n_lpot/2, shared_offset(n_lpot)*sizeof(T)>>>
        ((T*)device_values, (T*)(device_values + n_pitch), n);

    hipMemcpy(out, device_values + n_pitch, sizeof(T)*n, hipMemcpyDeviceToHost);
    hipFree(device_values);
}

int main()
{
    sranddev();

    static unsigned in_values[1024], out_values[1024];

    for (int i = 0; i < 1024; ++i)
        in_values[i] = rand() >> 21;

    prefix_sum(in_values, out_values, 1024);

    for (int i = 0; i < 1024; ++i)
        printf("%5d => %5d\n", in_values[i], out_values[i]);

    return 0;
}
